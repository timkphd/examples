#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
void checkCUDAError(const char *msg);
 __global__ void Kernel(int *dat);
main() {
	int *dat_local, *dat_remote;
	int gx,gy;
	int bx,by,bz;
	int size;
	int numthreads,j;
	
	printf(" %s\n","Enter grid dimensions: gx gy");
	scanf("%d %d",&gx,&gy);
	printf(" %s\n","Enter block dimensions: bx by bz");
	scanf("%d %d %d",&bx,&by,&bz);
	printf(" Grid dimensions:  %3d%4d\n",gx,gy);	
	printf(" Block dimensions: %3d%4d%4d\n",bx,by,bz);	
	dim3 dimGrid(gx,gy);
	dim3 dimBlock(bx,by,bz);
	
	numthreads=gx*gy*bx*by*bz;
	
	size=6*sizeof(int)*numthreads;
	hipMalloc((void**) &dat_remote, size);
        checkCUDAError("hipMalloc");
	dat_local=(int*)malloc(size);
	
	Kernel<<<dimGrid,dimBlock>>>(dat_remote);
        checkCUDAError("Kernel");
	hipMemcpy(dat_local, dat_remote, size,hipMemcpyDeviceToHost);
        checkCUDAError("copy");
	
	printf("%s\n","thread   blockid(x   y)   threadid(x   y   z)");
for(int i=0;i<numthreads;i++) {
		j=i*6;
		printf("%6d         %3d %3d           %3d %3d %3d\n",
		dat_local[j],
		dat_local[j+1],dat_local[j+2],
		dat_local[j+3],dat_local[j+4],dat_local[j+5]);
	}
			  
}


 __global__ void Kernel(int *dat) {
/* get my block within a grid */
    int myblock=blockIdx.x+blockIdx.y*gridDim.x;
/* how big is each block within a grid */
    int blocksize=blockDim.x*blockDim.y*blockDim.z;
/* get thread within a block */
    int subthread=threadIdx.z*(blockDim.x*blockDim.y)+threadIdx.y*blockDim.x+threadIdx.x;
/* find my thread */
    int thread=myblock*blocksize+subthread;
#if __DEVICE_EMULATION__
	printf("gridDim=(%3d %3d) blockIdx=(%3d %3d)     blockDim=(%3d %3d %3d)  threadIdx=(%3d %3d %3d)  %6d\n",    
	  gridDim.x,gridDim.y,
	  blockIdx.x,blockIdx.y,
	  blockDim.x,blockDim.y,blockDim.z,
	  threadIdx.x,threadIdx.y,threadIdx.z,thread);
#endif
/* starting index into array */
	int index=thread*6;
	dat[index]=thread;
	dat[index+1]=blockIdx.x;
	dat[index+2]=blockIdx.y;
	dat[index+3]=threadIdx.x;
	dat[index+4]=threadIdx.y;
	dat[index+5]=threadIdx.z;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

