#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "multShare.h"
// Thread block size
#define BLOCK_SIZE 16
 __global__ void MatMulKernel(const Matrix, const Matrix, Matrix);
/*
 * multShare.c
 *
 * Robert Hochberg
 * January 24, 2012
 *
 * Based nearly entirely on the code from the CUDA C Programming Guide
 */
// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = d_A.stride = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipError_t err = hipMalloc(&d_A.elements, size);
  printf("CUDA malloc A: %s\n",hipGetErrorString(err));
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = d_B.stride = B.width;
  d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  err = hipMalloc(&d_B.elements, size);
  printf("CUDA malloc B: %s\n",hipGetErrorString(err));
/* 37 */
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
  // Allocate C in device memory
  Matrix d_C;
  d_C.width = d_C.stride = C.width;
  d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  err = hipMalloc(&d_C.elements, size);
  printf("CUDA malloc C: %s\n",hipGetErrorString(err));
  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    err = hipDeviceSynchronize();
    printf("Run kernel: %s\n", hipGetErrorString(err));
  // Read C from device memory
  err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
  printf("Copy C off of device: %s\n",hipGetErrorString(err));
  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}
int main(int argc, char* argv[]){
  Matrix A, B, C;
  int a1, a2, b1, b2;

srand(1234);
  a1 = atoi(argv[1]); /* Height of A */
  a2 = atoi(argv[2]); /* Width  of A */
  b1 = a2;           /* Height of B */
/* 40 */

b2 = atoi(argv[3]); /* Width  of B */
A.height = a1;
A.width = a2;
A.elements = (float*)malloc(A.width * A.height * sizeof(float));
B.height = b1;
B.width = b2;
B.elements = (float*)malloc(B.width * B.height * sizeof(float));
C.height = A.height;
C.width = B.width;
C.elements = (float*)malloc(C.width * C.height * sizeof(float));
for(int i = 0; i < A.height; i++)
  for(int j = 0; j < A.width; j++)
    A.elements[i*A.width + j] = (rand() % 10);
for(int i = 0; i < B.height; i++)
  for(int j = 0; j < B.width; j++)
    B.elements[i*B.width + j] = (rand() % 5);
MatMul(A, B, C);
for(int i = 0; i < min(10, A.height); i++){
  for(int j = 0; j < min(10, A.width); j++)
    printf("%5.0f ", A.elements[i*A.width + j]);
  printf("\n");
}
printf("\n");
for(int i = 0; i < min(10, B.height); i++){
  for(int j = 0; j < min(10, B.width); j++)
    printf("%5.0f ", B.elements[i*B.width + j]);
  printf("\n");
}
printf("\n");
/* 41 */

for(int i = 0; i < min(10, C.height); i++){
  for(int j = 0; j < min(10, C.width); j++)
    printf("%5.0f ", C.elements[i*C.width + j]);
  printf("\n");
}
  printf("\n");
}

/* 42 */
