#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {
    int driver_version = 0, runtime_version = 0;

    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("Driver Version: %d\n"
           "Runtime Version: %d\n",
           driver_version, runtime_version);

    return 0;
}

