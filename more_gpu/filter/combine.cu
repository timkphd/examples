// nvc++   -O3 -static-nvidia -c++libs   combine.cu  -o reduce
// /usr/local/cuda-12.6/bin/nvcc   -DTRANSPOSE -O3 combine.cu -o treduce
// If copied to a container
// ml apptainer/1.1.9-ubkbfc2
// salloc --nodes=1 --time=04:00:00 --account=hpcapps --partition=gpu --exclusive --mem=0  --gres=gpu:4
// apptainer exec --nv  comp.sif /extra01/reduce


#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <stdio.h>
#include <float.h>
#include <limits.h>
#include <unistd.h>
#include <sys/time.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
        printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
        exit(0);                                                                            \
    }                                                                                     \
}while(0)


/* A gettimeofday routine to give access to the wall
   clock timer on most UNIX-like systems.  */
double mysecond()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp,&tzp);
    return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}


template <typename T>
__global__ void mytreduce(T const * __restrict__ const myorg, T * __restrict__ const b, int newspec, int oldspec, int newspac, int oldspac, int specfac, int spacfac,int len,int awidth)

{
    int nindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (nindex < len) {
            int newrow = nindex / newspec         ;
            int newcol = nindex - newrow * newspec ;
            int oldcol = newcol * specfac          ;
            int oldrow = newrow * spacfac          ;
            int oindex = oldrow * oldspec + oldcol ;
            float mysum=0;
            int count=0;
                for (int s=-awidth;s<=awidth;s++){
                    int i = oindex + s * oldspec;
                    if ((i > -1 ) && ((s + oldrow) < oldspac)){
                        mysum=mysum+myorg[i];
                        count++;
                    }
                }
                b[nindex]=(int)(((float)mysum)/count + 0.5);
    }
        
}


template <typename T>
__global__ void myreduce(T const * __restrict__ const myorg, T * __restrict__ const b, int newspec, int oldspec, int newspac, int oldspac, int specfac, int spacfac,int len,int awidth)

{
    int nindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (nindex < len) {
                int vnew = nindex/newspac;
                int hnew = nindex-vnew*newspac;
                int v = vnew*specfac;
                int h = hnew*spacfac;
                int oindex=v*oldspac+h;
                float mysum=0.0;
                int count=0;
                for (int s=-awidth;s<=awidth;s++){
                    int i=h+s;
                    if ((i > -1 ) && (i < oldspac)){
                        mysum=mysum+myorg[oindex+s];
                        count++;
                    }
                }
                b[nindex]=(int)(((float)mysum)/count + 0.5);
    }
        
}




int main(int argc, char** argv)
{
    int *d_a, *d_b, *A, *B;;

    int GPU=0;
    int N,OLD;
    int blockSize=192;
    hipSetDevice(GPU);
    
    int oldspec=1600;
    int oldspac=102;
    int w=10;
    int specfac = 1;
    int spacfac = 10;
    
// int rspec = Int16.Parse(args[0]);
// int rspac = Int16.Parse(args[1]);

       if (argc == 2){
           sscanf(argv[1],"%d",&spacfac);
       }
       if (argc == 3){
           sscanf(argv[1],"%d",&specfac);
           sscanf(argv[2],"%d",&spacfac);
       }
       if (argc == 4){
           sscanf(argv[1],"%d",&specfac);
           sscanf(argv[2],"%d",&spacfac);
           sscanf(argv[3],"%d",&w);
       }

    printf("specfac = %d    spacfac = %d    width = %d\n",specfac,spacfac,w);    
    int newspec=(oldspec/specfac);
    if ( (newspec*specfac) < oldspec ) newspec++;
    int newspac=(oldspac/spacfac);
    if ( (newspac*spacfac) < oldspac ) newspac++;
    printf("newspac %d    newspec %d\n",newspac,newspec);
    
    N=newspec*newspac;
    OLD=oldspec*oldspac;


    hipMalloc((void**)&d_a, sizeof(int)*OLD);
    hipMalloc((void**)&d_b, sizeof(int)*N);
    A=(int*)malloc(sizeof(int)*OLD);
    B=(int*)malloc(sizeof(int)*N);

    /* Compute execution configuration */
    dim3 dimBlock(blockSize);
    dim3 dimGrid(N/dimBlock.x );
    if( N % dimBlock.x != 0 ) dimGrid.x+=1;
    printf(" using %d threads per block, %d blocks\n",dimBlock.x,dimGrid.x);
    
    FILE *f18;
#ifdef TRANSPOSE
    f18=fopen("transpose","r");
#else
    f18=fopen("original","r");
#endif
    for (int ic=0;ic<OLD;ic++){
        fscanf(f18,"%d",&A[ic]);
    }
    fclose(f18);
    
    double tstart,tinn,tred,tout;
    tinn=0;
    tred=0;
    tout=0;
    

    int ntimes=1000;
    for (int icount=0;icount<ntimes;icount++){
        tstart=mysecond();
    /* Copy image to device */
        cudaErrorCheck( hipMemcpy(d_a, A, OLD*sizeof(int), hipMemcpyHostToDevice) );
        tinn=tinn+(mysecond()-tstart);
        
        tstart=mysecond();      
    /* shrink it */
#ifdef TRANSPOSE
        mytreduce<<<dimGrid,dimBlock>>>(d_a, d_b, newspec, oldspec, newspac, oldspac,specfac, spacfac,N,w);
#else
        myreduce<<<dimGrid,dimBlock>>>(d_a, d_b, newspec, oldspec, newspac, oldspac,specfac, spacfac,N,w);
#endif
        tred=tred+(mysecond()-tstart);
        
        tstart=mysecond();
    /* Copy image to cpu */
        cudaErrorCheck( hipMemcpy(B, d_b, N*sizeof(int), hipMemcpyDeviceToHost));
        tout=tout+(mysecond()-tstart);
    }
    printf("      INPUT           COMPUTE         OUTPUT\n");

    printf("%15g %15g %15g\n",tinn/ntimes,tred/ntimes,tout/ntimes);
    
     
#ifdef TRANSPOSE
    f18=fopen("new_gpu_transpose","w");
#else
    f18=fopen("new_gpu","w");
#endif

    for (int ic=0;ic<N;ic++){
        fprintf(f18,"%d\n",B[ic]);
    }
    fclose(f18);
    
    hipFree(d_a);
    hipFree(d_b);
    free(A);
    free(B);
}

